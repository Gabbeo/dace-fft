#include "hip/hip_runtime.h"
#include "../include/cufft_test.cuh"

#define N_TRANSFORMS 1

void cufft_float(std::vector<std::complex<float>*> inputs, std::vector<std::complex<float>*> outputs, std::vector<benchmark_info>& cufft_info, settings_s& settings) {
    size_t max_fft_length = *std::max_element(settings.FFT_LENGTHS.begin(), settings.FFT_LENGTHS.end());;
    
    hipfftComplex *in_data, *out_data;
    std::chrono::high_resolution_clock::time_point t1, t2;

    for (size_t i = 0; i < settings.FFT_LENGTHS.size(); i++) {
        size_t fft_length = settings.FFT_LENGTHS[i];
        double flop_count = 5 * fft_length * std::log2(fft_length);
        benchmark_info benchmark = benchmark_info(fft_length, flop_count);

        hipfftComplex* original_cufft_input = reinterpret_cast<hipfftComplex*>(inputs[i]);
        hipfftComplex* original_cufft_output = reinterpret_cast<hipfftComplex*>(outputs[i]);

        hipfftHandle plan;
        hipfftPlan1d(&plan, fft_length, HIPFFT_C2C, N_TRANSFORMS);

        hipMalloc((void**) &in_data, sizeof(hipfftComplex) * fft_length);
        hipMalloc((void**) &out_data, sizeof(hipfftComplex) * fft_length);
        hipfftExecC2C(plan, in_data, out_data, HIPFFT_FORWARD); // Warmup
        hipFree(in_data);
        hipFree(out_data);

        for (size_t iter = 0; iter < settings.NUM_ITERATIONS; iter++) {
            t1 = std::chrono::high_resolution_clock::now();
            hipMalloc((void**) &in_data, sizeof(hipfftComplex) * fft_length);
            hipMalloc((void**) &out_data, sizeof(hipfftComplex) * fft_length);
            hipMemcpy(in_data, original_cufft_input, fft_length * sizeof(hipfftComplex), hipMemcpyHostToDevice);
            hipfftExecC2C(plan, in_data, out_data, HIPFFT_FORWARD); 
            hipMemcpy(original_cufft_output, out_data, fft_length * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
            hipFree(in_data);
            hipFree(out_data);
            hipDeviceSynchronize();
            t2 = std::chrono::high_resolution_clock::now();
            double dur = std::chrono::duration_cast<std::chrono::duration<double>>(t2-t1).count();
            benchmark.add_timing(dur);
        }

        if (settings.DEBUG) {
            std::cout << "cuFFT, length: " << fft_length << ", time avg: " 
                << benchmark.t_avg() << "s, time sd: " << benchmark.t_sd() 
                << "s, flop/s avg: " << benchmark.flops_avg() << ", flop/s sd: "
                << benchmark.flops_sd() << "." << std::endl;
        } else {
            if (i % 50 == 0) {
                std::cout << "Completed step " << i << " of benchmarking"
                    << " cuFFT with fft_length " << fft_length << "." << std::endl;
            }
        }

        cufft_info.push_back(benchmark);

        hipfftDestroy(plan);
    }
}

void cufft_double(std::vector<std::complex<double>*> inputs, std::vector<std::complex<double>*> outputs, std::vector<benchmark_info>& cufft_info, settings_s& settings) {
    size_t max_fft_length = *std::max_element(settings.FFT_LENGTHS.begin(), settings.FFT_LENGTHS.end());;
    
    hipfftDoubleComplex *in_data, *out_data;
    std::chrono::high_resolution_clock::time_point t1, t2;

    for (size_t i = 0; i < settings.FFT_LENGTHS.size(); i++) {
        size_t fft_length = settings.FFT_LENGTHS[i];
        double flop_count = 5 * fft_length * std::log2(fft_length);
        benchmark_info benchmark = benchmark_info(fft_length, flop_count);

        hipfftDoubleComplex* original_cufft_input = reinterpret_cast<hipfftDoubleComplex*>(inputs[i]);
        hipfftDoubleComplex* original_cufft_output = reinterpret_cast<hipfftDoubleComplex*>(outputs[i]);

        hipfftHandle plan;
        hipfftPlan1d(&plan, fft_length, HIPFFT_Z2Z, N_TRANSFORMS);

        hipMalloc((void**) &in_data, sizeof(hipfftDoubleComplex) * fft_length);
        hipMalloc((void**) &out_data, sizeof(hipfftDoubleComplex) * fft_length);
        hipfftExecZ2Z(plan, in_data, out_data, HIPFFT_FORWARD); // Warmup
        hipFree(in_data);
        hipFree(out_data);

        for (size_t iter = 0; iter < settings.NUM_ITERATIONS; iter++) {
            t1 = std::chrono::high_resolution_clock::now();
            hipMalloc((void**) &in_data, sizeof(hipfftDoubleComplex) * fft_length);
            hipMalloc((void**) &out_data, sizeof(hipfftDoubleComplex) * fft_length);
            hipMemcpy(in_data, original_cufft_input, fft_length * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
            hipfftExecZ2Z(plan, in_data, out_data, HIPFFT_FORWARD); 
            hipMemcpy(original_cufft_output, out_data, fft_length * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
            hipFree(in_data);
            hipFree(out_data);
            hipDeviceSynchronize();
            t2 = std::chrono::high_resolution_clock::now();
            double dur = std::chrono::duration_cast<std::chrono::duration<double>>(t2-t1).count();
            benchmark.add_timing(dur);
        }

        if (settings.DEBUG) {
            std::cout << "cuFFT, length: " << fft_length << ", time avg: " 
                << benchmark.t_avg() << "s, time sd: " << benchmark.t_sd() 
                << "s, flop/s avg: " << benchmark.flops_avg() << ", flop/s sd: "
                << benchmark.flops_sd() << "." << std::endl;
        } else {
            if (i % 50 == 0) {
                std::cout << "Completed step " << i << " of benchmarking"
                    << " cuFFT with fft_length " << fft_length << "." << std::endl;
            }
        }

        cufft_info.push_back(benchmark);

        hipfftDestroy(plan);
    }

}